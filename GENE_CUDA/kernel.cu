#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <algorithm>
#include <vector>
using namespace std;
#define NS 10
#define ST 5//T�е����и��� TΪ�ο�ģʽ��
#define CNUM 4//Ⱥ���ģ ��ҪΪ2�ı��� ���ڷֳ�4�� ������64
#define NUM 5//��������
#define LEN 5
#define crossCNUM CNUM*(CNUM-1)
#define randomSize 500
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  
hipError_t geneWithCuda(int *crossseq[], int *seq[],int *ref[],double *fit);
class Chrom
{
public:
	Chrom() 
	{
		seq=new int[LEN];
		M=0;//�ǵ�Ҫ��0
		fit=0;
	}
	Chrom(const Chrom& a)
	{
		seq=new int[LEN];
		for(int i=0;i<LEN;i++)
		{
			seq[i]=a.seq[i];
		}
		M=a.M;
		fit=a.fit;
		//MM.assign(a.MM.begin(),a.MM.end());
	}
	~Chrom() {}
	int* seq ;
	//vector<int>MM;
	double M;
	double fit;//��Ӧֵ
};
bool Comp(Chrom& first,Chrom& second)
{
	return first.fit > second.fit;
}
typedef struct Code                           // �ṹ�����ͣ�Ϊ����Ⱦɫ��Ľṹ��
{
	int id;//0��Ԥ����*
	double count;
}code;     
__host__ void evpop (vector<Chrom>&popcurrent,vector<code>&codes,vector<int *>ref)   // ������������ɳ�ʼ��Ⱥ��
{
	int random ;
	for(int i=0;i<CNUM;i++)
	{
		Chrom chrom;
		for(int j=0;j<LEN;j++)
		{
			random=rand ()%NS;                     // ����һ�����ֵ
			chrom.seq[j]=codes[random].id;
		}
		popcurrent.push_back(chrom);
	}
	for(int i=0;i<CNUM;i++)
	{
		int random1=rand ()%3;
		for(int j=0;j<random1;j++)
		{
			random=rand ()%LEN; 
			popcurrent[i].seq[random]=0;
		}
	}
	for(int i=0;i<CNUM;i++)
	{
		for(int j=0;j<ST;j++)
		{
			bool eq=true;
			for(int k=0;k<LEN;k++)
			{
				int pop=popcurrent[i].seq[k];
				if(0!=pop&&ref[j][k]!=pop)
				{
					eq=false;
					break;
				}
			}
			if(eq)
			{
//				popcurrent[i].MM.push_back(j);
				popcurrent[i].M++;
			}
		}
	}
	for(int i=0;i<CNUM;i++)
	{
		int E=LEN;
		double ci=0;
		for(int j=0;j<LEN;j++)
		{
			if(popcurrent[i].seq[j]!=0)
			{
				ci+=codes[popcurrent[i].seq[j]-1].count;
			}
			else
			{
				E--;
			}
		}
		popcurrent[i].fit=ci*popcurrent[i].M*pow(NS,E)/ST;
		//cout<<"E: "<<E<<endl;
		//cout<<"Ns^E(Patterni)/S(T): "<<pow(NS,E)/ST<<endl;
		//cout<<"ci: "<<ci<<endl;
		//cout<<"M:"<<popcurrent[i].M<<endl;
		//cout<<"fit:    "<<popcurrent[i].fit<<endl;
	}             
}       
__host__ void pickchroms (vector<Chrom>& popcurrent,vector<Chrom>& popnext)          // ������ѡ����壻
{
	sort(popcurrent.begin(),popcurrent.end(),Comp);
	sort(popnext.begin(),popnext.end(),Comp);
	vector<Chrom> temp;
	int i=0,j=0;
	int nextlen=popnext.size();
	for(int k=0;k<CNUM;k++)
	{
		if(j>=nextlen||popcurrent[i].fit>popnext[j].fit)
		{
			Chrom ctemp(popcurrent[i]);
			temp.push_back(ctemp);
			i++;
		}
		else
		{
			Chrom ctemp(popnext[j]);
			temp.push_back(ctemp);
			j++;
		}
	}
	popnext.assign(temp.begin(),temp.end()); 
}   

__device__ int randomnum;
__device__  void calculate_fit(double ci,int E,int *seq,int **ref,double &fit)//����fitֵ
{
	int M=0;
	for(int k=0;k<ST;k++)
	{
		bool eq=true;
		for(int m=0;m<LEN;m++)
		{
			int refkm=ref[k][m];
			if(eq)
			{
				if(0!=seq[m]&&refkm!=seq[m])
				{
					eq=false;
					break;
				}
			}
		}
		if(eq)
		{
			M++;
		}
	}	
	fit=ci*M*_Pow_int(NS,E)/ST;
}
__global__ void crossover(int **crossseq,int **seq,int **ref,int* crosscount,int *random)//�������
{

	int x=CNUM/2*blockIdx.x+threadIdx.x;
	int y=CNUM/2*blockIdx.y+threadIdx.y;
	if(x>y)//xΪj,yΪi
	{	
		//printf("(%d,%d)\n",y,x); 
		//printf("(%d,%d)\n(%d)\n(%d)\n",y,x,seq[y][0],seq[x][0]); 
		int insert_pt=atomicAdd(crosscount, 2);
		int iscross=atomicAdd(&randomnum, 1);
		// printf("%d\n",random[insert_pt]);
		if(random[iscross]<80)
		{
			int crossloc=atomicAdd(&randomnum, 1);
			for(int i=0;i<random[crossloc]%LEN;i++)
			{
				// printf("%d\n",2*insert_pt);
				crossseq[insert_pt][i]=seq[y][i];
				crossseq[insert_pt+1][i]=seq[x][i];
			}
			for(int i=random[crossloc]%LEN;i<LEN;i++)
			{
				crossseq[insert_pt][i]=seq[x][i];
				crossseq[insert_pt+1][i]=seq[y][i];
			}
			//cross_calculate_fit(ci1,ci2,crossseq[insert_pt],crossseq[insert_pt+1],ref,fit[insert_pt],fit[insert_pt+1]);
		}
	}

}
__global__ void mutation(int **mutationseq,int **crossseq,int **ref,double*fit,int *random) // ���������
{
	int x=CNUM/2*blockIdx.x+threadIdx.x;
	double ci=0;
	int E=LEN;
	//printf("(%d)\n",x); 
	//printf("(%d,%d)\n(%d)\n(%d)\n",y,x,a[y][0],a[x][0]); 
	// printf("%d\n",random[insert_pt]);
	for(int i=0;i<LEN;i++)
	{
		int ismuta=atomicAdd(&randomnum ,1);
		if(random[ismuta]<5)//random<5�ĸ���Ϊ5%;
		{		
			int mutanum=atomicAdd(&randomnum ,1);
			mutationseq[x][i]=random[mutanum]%NS+1;
		}
		else
		{
			mutationseq[x][i]=crossseq[x][i];
		}
		if(mutationseq[x][i]!=0)
		{
			ci+=0.1;
		}
		else
		{
			E--;
		}
	}	
	calculate_fit(ci,E,mutationseq[x],ref,fit[x]);
}
int main()
{
	int* seq[CNUM] = {};
	int* crossseq[crossCNUM] = {};
	int *ref[ST]={};
	double *fit=new double[crossCNUM];
	for (int i = 0; i < CNUM; i++)
	{
		seq[i] =new int[LEN];
	}
	for (int i = 0; i < crossCNUM; i++)
	{
		crossseq[i]=new int[LEN];
	}
	for(int i=0;i<ST;i++)
	{
		ref[i]=new int [LEN];
	}
	seq[0][0]=1;
	seq[0][1]=2;
	seq[0][2]=0;
	seq[0][3]=4;
	seq[0][4]=5;
	//12045
	seq[1][0]=4;
	seq[1][1]=2;
	seq[1][2]=8;
	seq[1][3]=0;
	seq[1][4]=3;
	//42803
	seq[2][0]=5;
	seq[2][1]=2;
	seq[2][2]=1;
	seq[2][3]=4;
	seq[2][4]=7;
	//52147
	seq[3][0]=3;
	seq[3][1]=3;
	seq[3][2]=2;
	seq[3][3]=5;
	seq[3][4]=6;
	//33256


	ref[0][0]=1;
	ref[0][1]=2;
	ref[0][2]=3;
	ref[0][3]=4;
	ref[0][4]=7;
	//12347
	ref[1][0]=1;
	ref[1][1]=2;
	ref[1][2]=3;
	ref[1][3]=4;
	ref[1][4]=7;
	//12347
	ref[2][0]=5;
	ref[2][1]=2;
	ref[2][2]=3;
	ref[2][3]=4;
	ref[2][4]=5;
	//52345
	ref[3][0]=3;
	ref[3][1]=3;
	ref[3][2]=2;
	ref[3][3]=8;
	ref[3][4]=6;
	//33286
	ref[4][0]=1;
	ref[4][1]=2;
	ref[4][2]=2;
	ref[4][3]=5;
	ref[4][4]=6;
	//12256
	// Add vectors in parallel.
	hipError_t cudaStatus = geneWithCuda(crossseq, seq,ref,fit);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	for (int i=0;i<crossCNUM;i++)
	{
		for(int j=0;j<LEN;j++)
		{
			cout<<crossseq[i][j]<<" ";
		}
		cout<<endl;
		cout<<fit[i]<<endl;
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t geneWithCuda(int *outputseq[], int *seq[],int *ref[],double *fit)
{
	//srand(time(0));
	int *random=new int[randomSize];
	for(int i=0;i<randomSize;i++)
	{
		random[i]=rand()%100;    
	}
	int crosscount=0;//TODO:�ǵøĳ�0
	int *array_dev_input[CNUM] = {};
	int *array_dev_ref[ST]={};
	int *array_dev_crossoutput[crossCNUM] = {};
	int *array_dev_mutaoutput[crossCNUM] = {};
	int *dev_rand=0;
	double *dev_fit=0;
	int **dev_input = 0;
	int **dev_ref=0;
	int **dev_crossoutput = 0;
	int **dev_mutaoutput = 0;
	int *dev_crosscount;

	CHECK(hipSetDevice(0));
	CHECK(hipMalloc((void**)&(array_dev_input), CNUM * sizeof(int*)));
	for (int i = 0; i < CNUM; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_input[i]), LEN * sizeof(int)));
		CHECK(hipMemcpy(array_dev_input[i], seq[i], LEN * sizeof(int), hipMemcpyHostToDevice));
	}
	for (int i = 0; i < crossCNUM; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_crossoutput[i]), LEN * sizeof(int)));
		CHECK(hipMalloc((void**)&(array_dev_mutaoutput[i]), LEN * sizeof(int)));
	}
	for (int i = 0; i < ST; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_ref[i]), LEN * sizeof(int)));
		CHECK(hipMemcpy(array_dev_ref[i], ref[i], LEN * sizeof(int), hipMemcpyHostToDevice));
	}
	CHECK(hipMalloc((void**)&(dev_input), CNUM * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_crossoutput), crossCNUM * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_mutaoutput), crossCNUM * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_ref), ST * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_rand), randomSize * sizeof(int)));
	CHECK(hipMalloc((void**)&(dev_fit), crossCNUM * sizeof(double)));
	CHECK(hipMalloc((void**)&(dev_crosscount), sizeof(int)));

	CHECK(hipMemcpy(dev_input, array_dev_input, CNUM * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_crossoutput, array_dev_crossoutput, crossCNUM * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_mutaoutput, array_dev_mutaoutput, crossCNUM * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_ref, array_dev_ref, ST * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_crosscount, &crosscount, sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_rand, random, randomSize * sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_fit, fit, crossCNUM * sizeof(double), hipMemcpyHostToDevice));


	dim3 threads(CNUM/2,CNUM/2);
	dim3 blocks(2,2);
	crossover<<<blocks, threads>>>(dev_crossoutput,dev_input,dev_ref,dev_crosscount,dev_rand);
	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(&crosscount, dev_crosscount, sizeof(int), hipMemcpyDeviceToHost));
	mutation<<<1,crosscount>>>(dev_mutaoutput,dev_crossoutput,dev_ref,dev_fit,dev_rand);
	CHECK(hipDeviceSynchronize());
	for (int i = 0; i < crossCNUM; i++)
	{
		CHECK( hipMemcpy(outputseq[i], array_dev_crossoutput[i], LEN * sizeof(int), hipMemcpyDeviceToHost));
	}
	
	//for (int i = 0; i < crossCNUM; i++)
	//{
	//	CHECK( hipMemcpy(outputseq[i], array_dev_mutaoutput[i], LEN * sizeof(int), hipMemcpyDeviceToHost));
	//}
	CHECK(hipMemcpy(fit, dev_fit, crossCNUM*sizeof(double), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&crosscount, dev_crosscount, sizeof(int), hipMemcpyDeviceToHost));
}