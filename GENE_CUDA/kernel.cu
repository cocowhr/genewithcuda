#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
using namespace std;
#define NS 10
#define ST 5//T�е����и��� TΪ�ο�ģʽ��
#define CNUM 4//��ҪΪ2�ı��� ���ڷֳ�4�� ������64
#define LEN 5
#define crossCNUM CNUM*(CNUM-1)
#define randomSize 500
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  
hipError_t crossoverWithCuda(int *crossseq[], int *seq[],int *ref[],double *fit);
/*���ýṹ�� ֱ�Ӳ���*/
__device__ int randomnum;
__device__  void cross_calculate_fit(double ci1,double ci2,int *seq1,int *seq2,int **ref,double &fit1,double&fit2)
{
	int M1=0;
	int M2=0;
	for(int k=0;k<ST;k++)
	{
		bool eq1=true;
		bool eq2=true;
		for(int m=0;m<LEN;m++)
		{
			int refkm=ref[k][m];
			if(eq1)
			{
				if(0!=seq1[m]&&refkm!=seq1[m])
				{
					eq1=false;
				}
			}
			if(eq2)
			{
				if(0!=seq2[m]&&refkm!=seq2[m])
				{
					eq2=false;
				}
			}
			if(!eq1&&!eq2)
			{
				break;
			}
		}
		if(eq1)
		{
			M1++;
		}
		if(eq2)
		{
			M2++;
		}
	}	
	fit1=ci1*M1*_Pow_int(NS,LEN)/ST;
	fit2=ci2*M2*_Pow_int(NS,LEN)/ST;
}
__global__ void crossover(int **crossseq,int **seq,int **ref,double*fit,int* count,int *random)
{
	
	int x=CNUM/2*blockIdx.x+threadIdx.x;
	int y=CNUM/2*blockIdx.y+threadIdx.y;
	if(x>y)//xΪj,yΪi
	{	
		double ci1=0.5;
		double ci2=0.5;
		//printf("(%d,%d)\n",y,x); 
		//printf("(%d,%d)\n(%d)\n(%d)\n",y,x,a[y][0],a[x][0]); 
		int insert_pt=atomicAdd(count, 1);	
		int iscross=atomicAdd(&randomnum, 1);
		// printf("%d\n",random[insert_pt]);
		if(random[iscross]<80)
		{
			int crossloc=atomicAdd(&randomnum, 1);
			for(int i=0;i<random[crossloc]%LEN;i++)
			{
				// printf("%d\n",2*insert_pt);
				crossseq[2*insert_pt][i]=seq[y][i];
				crossseq[2*insert_pt+1][i]=seq[x][i];
			}
			for(int i=random[crossloc]%LEN;i<LEN;i++)
			{
				crossseq[2*insert_pt][i]=seq[x][i];
				crossseq[2*insert_pt+1][i]=seq[y][i];
			}
			cross_calculate_fit(ci1,ci2,crossseq[2*insert_pt],crossseq[2*insert_pt+1],ref,fit[2*insert_pt],fit[2*insert_pt+1]);
		}
	}

}
int main()
{
	int* seq[CNUM] = {};
	int* crossseq[crossCNUM] = {};
	int *ref[ST]={};
	double *fit=new double[crossCNUM];
	for (int i = 0; i < CNUM; i++)
	{
		seq[i] =new int[LEN];
	}
	for (int i = 0; i < crossCNUM; i++)
	{
		crossseq[i]=new int[LEN];
	}
	for(int i=0;i<ST;i++)
	{
		ref[i]=new int [LEN];
	}
	seq[0][0]=1;
	seq[0][1]=2;
	seq[0][2]=3;
	seq[0][3]=4;
	seq[0][4]=5;
	//12345
	seq[1][0]=4;
	seq[1][1]=2;
	seq[1][2]=8;
	seq[1][3]=6;
	seq[1][4]=3;
	//42863
	seq[2][0]=5;
	seq[2][1]=2;
	seq[2][2]=1;
	seq[2][3]=4;
	seq[2][4]=7;
	//52147
	seq[3][0]=3;
	seq[3][1]=3;
	seq[3][2]=2;
	seq[3][3]=5;
	seq[3][4]=6;
	//33256


	ref[0][0]=1;
	ref[0][1]=2;
	ref[0][2]=1;
	ref[0][3]=4;
	ref[0][4]=7;
	//12147
	ref[1][0]=1;
	ref[1][1]=2;
	ref[1][2]=1;
	ref[1][3]=4;
	ref[1][4]=7;
	//12147
	ref[2][0]=5;
	ref[2][1]=2;
	ref[2][2]=3;
	ref[2][3]=4;
	ref[2][4]=5;
	//52345
	ref[3][0]=3;
	ref[3][1]=3;
	ref[3][2]=2;
	ref[3][3]=8;
	ref[3][4]=6;
	//33286
	ref[4][0]=1;
	ref[4][1]=2;
	ref[4][2]=2;
	ref[4][3]=5;
	ref[4][4]=6;
	//12256
	// Add vectors in parallel.
	hipError_t cudaStatus = crossoverWithCuda(crossseq, seq,ref,fit);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	for (int i=0;i<crossCNUM;i++)
	{
		for(int j=0;j<LEN;j++)
		{
			cout<<crossseq[i][j]<<" ";
		}
		cout<<endl;
		cout<<fit[i]<<endl;
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t crossoverWithCuda(int *crossseq[], int *seq[],int *ref[],double *fit)
{
	//srand(time(0));
	int *random=new int[randomSize];
	for(int i=0;i<randomSize;i++)
	{
		random[i]=rand()%100;    
	}
	int count=0;
	int *array_dev_input[CNUM] = {};
	int *array_dev_ref[ST]={};
	int *array_dev_output[crossCNUM] = {};
	int *dev_rand=0;
	double *dev_fit=0;
	int **dev_input = 0;
	int **dev_ref=0;
	int **dev_output = 0;
	int *dev_count;
	CHECK(hipSetDevice(0));
	CHECK(hipMalloc((void**)&(array_dev_input), CNUM * sizeof(int*)));
	for (int i = 0; i < CNUM; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_input[i]), LEN * sizeof(int)));
		CHECK(hipMemcpy(array_dev_input[i], seq[i], LEN * sizeof(int), hipMemcpyHostToDevice));
	}
	for (int i = 0; i < crossCNUM; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_output[i]), LEN * sizeof(int)));
	}
	for (int i = 0; i < ST; i++)
	{
		CHECK(hipMalloc((void**)&(array_dev_ref[i]), LEN * sizeof(int)));
		CHECK(hipMemcpy(array_dev_ref[i], ref[i], LEN * sizeof(int), hipMemcpyHostToDevice));
	}
	CHECK(hipMalloc((void**)&(dev_input), CNUM * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_output), crossCNUM * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_ref), ST * sizeof(int*)));
	CHECK(hipMalloc((void**)&(dev_rand), randomSize * sizeof(int)));
	CHECK(hipMalloc((void**)&(dev_fit), crossCNUM * sizeof(double)));
	CHECK(hipMalloc((void**)&(dev_count), sizeof(int)));
	CHECK(hipMemcpy(dev_input, array_dev_input, CNUM * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_output, array_dev_output, crossCNUM * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_ref, array_dev_ref, ST * sizeof(int*), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_rand, random, randomSize * sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_fit, fit, crossCNUM * sizeof(double), hipMemcpyHostToDevice));
	dim3 threads(CNUM/2,CNUM/2);
	dim3 blocks(2,2);
	crossover<<<blocks, threads>>>(dev_output,dev_input,dev_ref,dev_fit,dev_count,dev_rand);
	CHECK(hipDeviceSynchronize());
	for (int i = 0; i < crossCNUM; i++)
	{
		CHECK( hipMemcpy(crossseq[i], array_dev_output[i], LEN * sizeof(int), hipMemcpyDeviceToHost));
	}
		CHECK(hipMemcpy(fit, dev_fit, crossCNUM*sizeof(double), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost));
}